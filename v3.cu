#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState *state) {
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state, int n, unsigned int *result) {
    int id = threadIdx.x + blockIdx.x * 64;
    int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random unsigned ints */
    for(int i = 0; i < n; i++) {
        x = hiprand(&localState);
	printf("%d ", x);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_uniform_kernel(hiprandState *state, int n, unsigned int *result) {
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int i = 0; i < n; i++) {
        x = hiprand_uniform(&localState);
        /* Check if > .5 */
        if(x > .5) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_normal_kernel(hiprandState *state, int n, unsigned int *result) {
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float2 x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random normals */
    for(int i = 0; i < n/2; i++) {
        x = hiprand_normal2(&localState);
        /* Check if within one standard deviaton */
        if((x.x > -1.0) && (x.x < 1.0)) {
            count++;
        }
        if((x.y > -1.0) && (x.y < 1.0)) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}


int main(int argc, char *argv[]) {
    int i;
    unsigned int total;
    hiprandState *devStates;
    unsigned int *devResults, *hostResults;
    int sampleCount = 10000;
    
    /* Allocate space for results on host */
    hostResults = (unsigned int *)calloc(64 * 64, sizeof(int));

    /* Allocate space for results on device */
    hipMalloc((void **)&devResults, 64 * 64 * sizeof(unsigned int));

    /* Set results to 0 */
    hipMemset(devResults, 0, 64 * 64 * sizeof(unsigned int));

    hipMalloc((void **)&devStates, 64 * 64 * sizeof(hiprandState));
    setup_kernel<<<64, 64>>>(devStates);
    generate_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
    
    /* Copy device memory to host */
    hipMemcpy(hostResults, devResults, 64 * 64 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    //printf("%10.13f\n", (float)total / (64.0f * 64.0f * sampleCount * 50.0f));
        
    /* Set results to 0 */
    hipMemset(devResults, 0, 64 * 64 * sizeof(unsigned int));

    /* Generate and use uniform pseudo-random  */
    generate_uniform_kernel<<<64, 64>>>(devStates, sampleCount, devResults);

    /* Copy device memory to host */
    hipMemcpy(hostResults, devResults, 64 * 64 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    //printf("%10.13f\n", (float)total / (64.0f * 64.0f * sampleCount * 50.0f));
    
    /* Set results to 0 */
    hipMemset(devResults, 0, 64 * 64 * sizeof(unsigned int));

    /* Generate and use normal pseudo-random  */
    generate_normal_kernel<<<64, 64>>>(devStates, sampleCount, devResults);

    /* Copy device memory to host */
    hipMemcpy(hostResults, devResults, 64 * 64 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    //printf("%10.13f\n", (float)total / (64.0f * 64.0f * sampleCount * 50.0f));

    /* Cleanup */
    hipFree(devStates);
    hipFree(devResults);
    free(hostResults);
    return EXIT_SUCCESS;
}